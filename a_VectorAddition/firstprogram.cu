
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <cassert>
#include <iostream>

using namespace std;

__global__ void vectorAdd(int *a, int*b, int*c , int N){
    
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid<N){
        c[tid]=a[tid]+b[tid];
    }


}

void init_array(int *a, int N){
    for (int i=0;i<N;i++){
        a[i] = rand() % 100;
    }
}

void verifyAdd(int *a, int*b, int*c , int N){
    for(int i=0;i<N;i++){
        assert(a[i]+b[i]==c[i]);
    }
}

int main(){


    int N = 1 << 20;
    size_t bytes = N * sizeof(bytes);

    int* a, * b, * c;

    hipMallocManaged(&a,bytes);
    hipMallocManaged(&b,bytes);
    hipMallocManaged(&c,bytes);


    init_array(a, N);
    init_array(b, N);


    int THREADS = 256;
    int BLOCKS  = (N+THREADS-1)/THREADS;


    vectorAdd<<<BLOCKS,THREADS>>>(a,b,c,N);
    hipDeviceSynchronize();

    verifyAdd(a,b,c,N);

    cout << "YAAAY" << endl;
    
    return 0;
}